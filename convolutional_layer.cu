#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "layer.hpp"

#include <iostream>

__global__ static void Cuda_Convolutional_Layer_Forward_Pass(double* batched_inputs, double* weights, double* bias, double* forward_output, 
						size_t batch_size, size_t kernals,size_t kernal_size, size_t padding, size_t output_size, size_t channels, size_t input_size, size_t stride) {
	
	size_t batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
	size_t kernal_idx = blockIdx.y * blockDim.y + threadIdx.y;
	size_t position_idx = blockIdx.z * blockDim.z + threadIdx.z;
	
	if (batch_idx < batch_size && kernal_idx < kernals && position_idx < (output_size * output_size)) {
		
		size_t neurons = output_size * output_size * kernals;
		forward_output[batch_idx * neurons + kernal_idx * output_size * output_size + position_idx] = bias[kernal_idx];
		int starting_y_pos = (position_idx / output_size) * stride - padding;
		int starting_x_pos = (position_idx % output_size) * stride - padding;
		
		for (int y = 0; y < (int)kernal_size; y++) {

			if (starting_y_pos + y < 0) {
				continue;
			}

			if (starting_y_pos + y >= input_size) {
				break;
			}

			for (int x = 0; x < (int)kernal_size; x++) {

				if (starting_x_pos + x < 0){ 
					continue;
				}
				if (starting_x_pos + x >= input_size) {
					break;
				}
				for (int z = 0; z < (int)channels; z++) {
					forward_output[batch_idx * neurons + kernal_idx * output_size * output_size + position_idx] += weights[kernal_idx * channels * kernal_size * kernal_size + (z * kernal_size * kernal_size) + (y * kernal_size) + x] * batched_inputs[batch_idx * channels * input_size * input_size + z * input_size * input_size + (starting_y_pos + y) * input_size + (starting_x_pos + x)];
				}
			}
		}
	}
}

__global__ static void Cuda_Convolutional_Layer_Init_Back_Propigation(double* batched_targets, double* forward_ouput, double* backward_input, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		backward_input[batch_idx * neurons + neuron_idx] = 2.0 * (forward_ouput[batch_idx * neurons + neuron_idx] - batched_targets[batch_idx * neurons + neuron_idx]) / (double)(batch_size * neurons);
	}
}

__global__ static void Cuda_Sigmoid_Activation_Forward_Pass(double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		forward_output[batch_idx * neurons + neuron_idx] = 1.0 / (1.0 + std::powf(2.71828182846, -1.0 * forward_output[batch_idx * neurons + neuron_idx]));
	}
}

__global__ static void Cuda_Sigmoid_Activation_Backward_Pass(double* backward_input, double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		backward_input[batch_idx * neurons + neuron_idx] *= forward_output[batch_idx * neurons + neuron_idx] * (1.0 - forward_output[batch_idx * neurons + neuron_idx]);
	}
}

__global__ static void Cuda_Rectified_Linear_Activation_Forward_Pass(double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons && forward_output[batch_idx * neurons + neuron_idx] < 0.0) {
		forward_output[batch_idx * neurons + neuron_idx] = 0.0;
	}
}

__global__ static void Cuda_Rectified_Linear_Activation_Backward_Pass(double* backward_input, double* forward_input, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons && forward_input[batch_idx * neurons + neuron_idx] == 0.0) {
		backward_input[batch_idx * neurons + neuron_idx] = 0.0;
	}
}

convolutional_layer::convolutional_layer() { 
	weights = nullptr;
	bias = nullptr;
	d_weights = nullptr;
	d_bias = nullptr;
	forward_output = nullptr;
	backward_input = nullptr;
	kernals = 0;
	kernal_size = 0;
	channels = 0;
	padding = 0;
	stride = 0;
	input_size = 0;
	output_size = 0;
	batch_size = 0;
	neurons = 0;
	inputs = 0;
}

convolutional_layer::convolutional_layer(size_t _input_size, size_t _channels, size_t _kernals, size_t _kernal_size, size_t _stride, size_t _padding) {

	kernals = _kernals;
	kernal_size = _kernal_size;
	channels = _channels;
	padding = _padding;
	stride = _stride;
	input_size = _input_size;
	batch_size = 0;
	
	output_size = (input_size + (2 * padding) - kernal_size) / stride + 1;
	neurons = output_size * output_size * kernals;
	inputs = input_size * input_size * channels;

	weights = (double*)malloc(kernal_size * kernal_size * channels * kernals * sizeof(double));
	d_weights = (double*)malloc(kernal_size * kernal_size * channels * kernals * sizeof(double));
	bias = (double*)malloc(kernals * sizeof(double));
	d_bias = (double*)malloc(kernals * sizeof(double));

	forward_output = nullptr;
	backward_input = nullptr;
	layer_activation_function = activation_functions::Linear;

	if (weights == nullptr || d_weights == nullptr || bias == nullptr || d_bias == nullptr) {
		std::cerr << "Error: Could not allocate memory in convolutional layer" << std::endl;
		exit(EXIT_FAILURE);
	}  

	for (size_t i = 0; i < kernals; i++) {
		bias[i] = (double)i;
		for (size_t j = 0; j < channels * kernal_size * kernal_size; j++) {
			weights[i * kernal_size * kernal_size * channels + j] = (double)(i * kernal_size * kernal_size * channels + j);
		}
	}	
}

convolutional_layer::~convolutional_layer() {
	free(weights);
	free(d_weights);
	free(bias);
	free(d_bias);
	free(forward_output);
	free(backward_input);
}

void convolutional_layer::forward(const std::vector<std::vector<double>>& batched_inputs) {
	std::cerr << "Error: Auto de-flattening not supported at this time" << std::endl;
	exit(EXIT_FAILURE);
}

void convolutional_layer::forward(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_inputs) {
	
	double* input_arr = (double*)malloc(batched_inputs.size() * inputs * sizeof(double));
	
	if (input_arr == nullptr) {
		std::cerr << "Error: Could not allocate memory in convolutional_layer for forward pass" << std::endl;
		exit(EXIT_FAILURE);
	}

	for (size_t i = 0; i < batched_inputs.size(); i++) {
		if (batched_inputs[i].size() != channels) {
			std::cerr << "Error: Inputs incomptibale shape with convolutional layer" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (size_t j = 0; j < channels; j++) {

			if (batched_inputs[i][j].size() != input_size) {
				std::cerr << "Error: Inputs incomptibale shape with convolutional layer" << std::endl;
				exit(EXIT_FAILURE);
			}

			for (size_t z = 0; z < input_size; z++) {

				if (batched_inputs[i][j][z].size() != input_size) {
					std::cerr << "Error: Inputs incomptibale shape with convolutional layer" << std::endl;
					exit(EXIT_FAILURE);
				}

				memcpy(input_arr + z * input_size + j * input_size * input_size + i * channels * input_size * input_size, batched_inputs[i][j][z].data(), input_size * sizeof(double));
			}
		}
	}
	forward(input_arr, inputs, batched_inputs.size());

	free(input_arr);
}

void convolutional_layer::forward(const layer* prev_layer) {

	if (prev_layer->neurons != inputs) {
		std::cerr << "Error: convolutional_layer of incomptibale shape with connected layer" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	forward(prev_layer->forward_output, prev_layer->neurons, prev_layer->batch_size);
}

void convolutional_layer::forward(double* batched_inputs, size_t _input_size, size_t _batch_size) {
	
	if (inputs != _input_size) {
		std::cerr << "Error: Incompatible input shape with convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (batch_size != _batch_size) {
		if (forward_output != nullptr) free(forward_output);
		forward_output = (double*)malloc(_batch_size * neurons * sizeof(double));
		
		if (backward_input != nullptr) {
			free(backward_input);
			backward_input = nullptr;
		}

		if (forward_output == nullptr) {
			std::cerr << "Error: Could not allocate memory in convolutional layer for forward pass" << std::endl;
			exit(EXIT_FAILURE);
		}

		batch_size = _batch_size;
	}

	double* cuda_batched_inputs = nullptr;
	double* cuda_weights = nullptr;
	double* cuda_bias = nullptr;
	double* cuda_forward_output = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&cuda_batched_inputs, batch_size * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_weights, kernals * channels * kernal_size * kernal_size * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_bias, kernals * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_forward_output, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_batched_inputs, batched_inputs, batch_size * inputs * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_weights, weights, kernals * channels * kernal_size * kernal_size * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_bias, bias, kernals * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	dim3 blocks(batch_size/6 + 1, kernals/6 + 1, (output_size * output_size)/6 + 1);
	dim3 threads(6, 6, 6);
	Cuda_Convolutional_Layer_Forward_Pass<<<blocks, threads>>>(cuda_batched_inputs, cuda_weights, cuda_bias, cuda_forward_output, batch_size, kernals, kernal_size, padding, output_size, channels, input_size, stride);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: failed to launch convolutional layer forward pass kernal" << std::endl;
		exit(error_code);
	}

	if (layer_activation_function == activation_functions::Sigmoid) {
		dim3 blocks_2d(4, 4);
		dim3 threads_2d(16, 16);
		Cuda_Sigmoid_Activation_Forward_Pass<<<blocks_2d, threads_2d>>>(cuda_forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Rectified_Linear) {
		dim3 blocks_2d(4, 4);
		dim3 threads_2d(16, 16);
		Cuda_Rectified_Linear_Activation_Forward_Pass<<<blocks_2d, threads_2d>>>(cuda_forward_output, batch_size, neurons);
	}
	
	if (layer_activation_function != activation_functions::Linear && (error_code = hipGetLastError()) != hipError_t::hipSuccess) {
		std::cerr << "Error: failed to launch convolutional layer forward activation function kernal" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(forward_output, cuda_forward_output, batch_size * neurons * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to host failed" << std::endl;
		exit(error_code);
	}

	hipFree(cuda_batched_inputs);
	hipFree(cuda_weights);
	hipFree(cuda_bias);
	hipFree(cuda_forward_output);
}

double convolutional_layer::loss(const std::vector<std::vector<double>>& batched_targets) const {

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Incompatible batch size, cannot calculate loss in convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	double result = 0.0;

	for (size_t i = 0; i < batch_size; i++) {
		
		if (batched_targets[i].size() != neurons) {
			std::cerr << "Error: Incompatible input shape, cannot calculate loss in convolutional_layer" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (size_t j = 0; j < neurons; j++) {
			result += ((forward_output[i * neurons + j] - batched_targets[i][j]) * (forward_output[i * neurons + j] - batched_targets[i][j])) / (double)(neurons * batch_size);
		}
	}
	
	return result;
}
double convolutional_layer::loss(const std::vector<int>& batched_targets) const {

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Incompatible batch size, cannot calculate loss in convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	//need to check my softmax math here.
	return 0.0;
}
double convolutional_layer::loss(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_targets) const {

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Incompatible batch size, cannot calculate loss in convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	double result = 0.0;
	
	for (size_t i = 0; i < batch_size; i++) {

		if (batched_targets[i].size() != kernals) {
			std::cerr << "Error: Incompatible input shape, cannot calculate loss in convolutional_layer" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (size_t j = 0; j < channels; j++) {

			if (batched_targets[i][j].size() != output_size) {
				std::cerr << "Error: Incompatible input shape, cannot calculate loss in convolutional_layer" << std::endl;
				exit(EXIT_FAILURE);
			}

			for (size_t y = 0; y < output_size; y++) {

				if (batched_targets[i][j][y].size() != output_size) {
					std::cerr << "Error: Incompatible input shape, cannot calculate loss in convolutional_layer" << std::endl;
					exit(EXIT_FAILURE);
				}
				
				for (size_t x = 0; x < output_size; x++) {
					result += ((forward_output[i * neurons + j * output_size * output_size + y * output_size + x] - batched_targets[i][j][y][x]) * (forward_output[i * neurons + j * output_size * output_size + y * output_size + x] - batched_targets[i][j][y][x])) / (double)(neurons * batch_size);	
				}
			}
		}
	}

	return result;
}

void convolutional_layer::init_back_propigation(const std::vector<std::vector<double>>& batched_targets) {

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* input_arr = (double*)malloc(batch_size * neurons * sizeof(double));

	if (input_arr == nullptr) {
		std::cerr << "Error: Could not allocted memory in convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	for (size_t i = 0; i < batch_size; i++) {
		
		if (batched_targets[i].size() != neurons) {
			std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
			exit(EXIT_FAILURE);
		}

		memcpy(input_arr + i * batch_size, batched_targets[i].data(), neurons * sizeof(double));
	}

	init_back_propigation(input_arr, neurons, batch_size);
}
void convolutional_layer::init_back_propigation(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_targets) {

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* input_arr = (double*)malloc(batch_size * neurons * sizeof(double));

	if (input_arr == nullptr) {
		std::cerr << "Error: Could not allocted memory in convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}


	for (size_t i = 0; i < batch_size; i++) {

		if (batched_targets[i].size() != kernals) {
			std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (size_t j = 0; j < kernals; j++) {

			if (batched_targets[i][j].size() != output_size) {
				std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
				exit(EXIT_FAILURE);
			}

			for (size_t y = 0; y < output_size; y++) {
				
				if (batched_targets[i][j][y].size() != output_size) {
					std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
					exit(EXIT_FAILURE);
				}

				memcpy(input_arr + i * neurons + j * output_size * output_size + y * output_size, batched_targets[i][j][y].data(), output_size * sizeof(double));
			}
		}
	}

	init_back_propigation(input_arr, neurons, batch_size);
}
void convolutional_layer::init_back_propigation(double* batched_targets, size_t _input_size, size_t _batch_size) {

	if (_input_size != neurons || _batch_size != batch_size) {
		std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (backward_input == nullptr) {
		backward_input = (double*)malloc(batch_size * neurons * sizeof(double));
		if (backward_input == nullptr) {
			std::cerr << "Error: Unable to allocate memory for backpropigation in convolutional_layer" << std::endl;
			exit(EXIT_FAILURE);
		}
	}

	double* cuda_forward_output = nullptr;
	double* cuda_batched_targets = nullptr;
	double* cuda_backward_input = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&cuda_forward_output, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_batched_targets, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_backward_input, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_forward_output, forward_output, batch_size * neurons * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_batched_targets, batched_targets, batch_size * neurons * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	dim3 blocks(neurons / 16 + 1, batch_size / 16 + 1);
	dim3 threads(16, 16);
	
	Cuda_Convolutional_Layer_Init_Back_Propigation<<<blocks, threads>>>(cuda_batched_targets, cuda_forward_output, cuda_backward_input, batch_size, neurons);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch init back propigation kernal in convolutional_layer" << std::endl;
		exit(error_code);
	}

	//activation function calls go here.
	if (layer_activation_function == activation_functions::Sigmoid) {
		Cuda_Sigmoid_Activation_Backward_Pass<<<blocks, threads>>>(cuda_backward_input, cuda_forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Rectified_Linear) {
		Cuda_Rectified_Linear_Activation_Backward_Pass<<<blocks, threads>>>(backward_input, forward_output, batch_size, neurons);
	}

	if (layer_activation_function != activation_functions::Linear && (error_code = hipGetLastError()) != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch backward activation function kernal" << std::endl;
		exit(error_code);
	}
	
	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}
	
	error_code = hipMemcpy(backward_input, cuda_backward_input, batch_size * neurons * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to host failed" << std::endl;
		exit(error_code);
	}

	hipFree(cuda_forward_output);
	hipFree(cuda_batched_targets);
	hipFree(cuda_backward_input);
}










