#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "layer.hpp"

#include <random>
#include <iostream>

__global__ static void Cuda_Convolutional_Layer_Forward_Pass(double* batched_inputs, double* weights, double* bias, double* forward_output, 
						size_t batch_size, size_t kernals,size_t kernal_size, size_t padding, size_t output_size, size_t channels, size_t input_size, size_t stride) {
	
	size_t batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
	size_t kernal_idx = blockIdx.y * blockDim.y + threadIdx.y;
	size_t position_idx = blockIdx.z * blockDim.z + threadIdx.z;
	
	if (batch_idx < batch_size && kernal_idx < kernals && position_idx < (output_size * output_size)) {
		
		size_t neurons = output_size * output_size * kernals;
		forward_output[batch_idx * neurons + kernal_idx * output_size * output_size + position_idx] = bias[kernal_idx];
		int starting_y_pos = (position_idx / output_size) * stride - padding;
		int starting_x_pos = (position_idx % output_size) * stride - padding;
		
		for (int y = 0; y < (int)kernal_size; y++) {

			if (starting_y_pos + y < 0) {
				continue;
			}

			if (starting_y_pos + y >= input_size) {
				break;
			}

			for (int x = 0; x < (int)kernal_size; x++) {

				if (starting_x_pos + x < 0){ 
					continue;
				}
				if (starting_x_pos + x >= input_size) {
					break;
				}
				for (int z = 0; z < (int)channels; z++) {
					forward_output[batch_idx * neurons + kernal_idx * output_size * output_size + position_idx] += weights[kernal_idx * channels * kernal_size * kernal_size + (z * kernal_size * kernal_size) + (y * kernal_size) + x] * batched_inputs[batch_idx * channels * input_size * input_size + z * input_size * input_size + (starting_y_pos + y) * input_size + (starting_x_pos + x)];
				}
			}
		}
	}
}

__global__ static void Cuda_Convolutional_Layer_First_Backward_Pass(double* batched_inputs, double* backward_input, double* d_weights, 
									size_t batch_size, size_t kernals, size_t channels, size_t kernal_size, size_t input_size, size_t output_size, size_t stride, size_t padding) {

	size_t kernal_idx = blockIdx.x * blockDim.x + threadIdx.x;
	size_t channel_idx = blockIdx.y * blockDim.y + threadIdx.y;
	size_t position_idx = blockIdx.z * blockDim.z + threadIdx.z;

	if (kernal_idx < kernals && channel_idx < channels && position_idx < (kernal_size * kernal_size)) {

		int idx = kernal_idx * channels * kernal_size * kernal_size + channel_idx * kernal_size * kernal_size + position_idx;
		d_weights[idx] = 0.0;

		int weight_y_offset = position_idx / (int)kernal_size;
		int weight_x_offset = position_idx % (int)kernal_size;
			
		for (int y = 0; y < (int)output_size; y++) {

			for (int x = 0; x < (int)output_size; x++) {

				int position_y = (y * (int)stride) - (int)padding + weight_y_offset;
				int position_x = (x * (int)stride) - (int)padding + weight_x_offset;

				if (position_y < 0 || position_y >= input_size || position_x < 0 || position_x >= input_size) {
					continue;
				}
				
				for (int i = 0; i < (int)batch_size; i++) {
					d_weights[idx] += backward_input[i * kernals * output_size * output_size + kernal_idx * output_size * output_size + y * output_size + x] * batched_inputs[i * channels * input_size * input_size + channel_idx * input_size * input_size + position_y * input_size + position_x];
				}
			}
		}
	}
}

__global__ static void Cuda_Convolutional_Layer_Second_Backward_Pass(double* backward_input, double* d_bias, size_t batch_size, size_t kernals, size_t output_size) {
	
	size_t kernal_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (kernal_idx < kernals) {

		d_bias[kernal_idx] = 0.0;

		for (size_t i = 0; i < batch_size; i++) {
			for (size_t j = 0; j < output_size * output_size; j++) {
				d_bias[kernal_idx] += backward_input[i * output_size * output_size * kernals + kernal_idx * output_size * output_size + j];
			}
		}
	}
}

__global__ static void Cuda_Convolutional_Layer_Init_Back_Propigation(double* batched_targets, double* forward_ouput, double* backward_input, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		backward_input[batch_idx * neurons + neuron_idx] = 2.0 * (forward_ouput[batch_idx * neurons + neuron_idx] - batched_targets[batch_idx * neurons + neuron_idx]) / (double)(batch_size * neurons);
	}
}

__global__ static void Cuda_Convolution_Layer_Partial_Derivitive_of_Loss(double* backward_input, double* weights, double* prev_layer_backward_input,
												size_t batch_size, size_t kernals, size_t channels, size_t kernal_size, size_t input_size, size_t padding, size_t stride, size_t output_size) {
	
	size_t batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
	size_t channel_idx = blockIdx.y * blockDim.y + threadIdx.y;
	size_t position_idx = blockIdx.z * blockDim.z + threadIdx.z;

	if (batch_idx < batch_size && channel_idx < channels && position_idx < (input_size * input_size)) {

		int idx = batch_idx * channels * input_size * input_size + channel_idx * input_size * input_size + position_idx;
		int position_y_idx = position_idx / input_size;
		int position_x_idx = position_idx % input_size;
		prev_layer_backward_input[idx] = 0.0;
		
		for (int y = 0; y < kernal_size; y++) {
			
			if ((int)(position_y_idx - y + (int)padding) < 0 || position_y_idx + (int)kernal_size - 1 - y >= (int)input_size + (int)padding ||(int)(position_y_idx - y + (int)padding) % stride != 0) {
				continue;
			}
			
			for (int x = 0; x < kernal_size; x++) {

				if ((int)(position_x_idx - x + (int)padding) < 0 || position_x_idx + (int)kernal_size - 1 - x >= (int)input_size + (int)padding ||(int)(position_x_idx - x + (int)padding) % stride != 0) {
					continue;
				}

				int output_y = (position_y_idx - y + padding) / stride;
				int output_x = (position_x_idx - x + padding) / stride;
				
				for (int i = 0; i < kernals; i++) {
					prev_layer_backward_input[idx] += weights[i * channels * kernal_size * kernal_size + channel_idx * kernal_size * kernal_size + y * kernal_size + x] * backward_input[batch_idx * kernals * output_size * output_size + i * output_size * output_size + output_y * output_size + output_x];
				}	
			}
		}
	}
}

__global__ static void Cuda_Sigmoid_Activation_Forward_Pass(double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		forward_output[batch_idx * neurons + neuron_idx] = 1.0 / (1.0 + std::powf(2.71828182846, -1.0 * forward_output[batch_idx * neurons + neuron_idx]));
	}
}

__global__ static void Cuda_Sigmoid_Activation_Backward_Pass(double* backward_input, double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		backward_input[batch_idx * neurons + neuron_idx] *= forward_output[batch_idx * neurons + neuron_idx] * (1.0 - forward_output[batch_idx * neurons + neuron_idx]);
	}
}

__global__ static void Cuda_Rectified_Linear_Activation_Forward_Pass(double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons && forward_output[batch_idx * neurons + neuron_idx] < 0.0) {
		forward_output[batch_idx * neurons + neuron_idx] = 0.0;
	}
}

__global__ static void Cuda_Rectified_Linear_Activation_Backward_Pass(double* backward_input, double* forward_input, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons && forward_input[batch_idx * neurons + neuron_idx] == 0.0) {
		backward_input[batch_idx * neurons + neuron_idx] = 0.0;
	}
}

__global__ static void Cuda_Matrix_Addition(double* residual_inputs, double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		forward_output[batch_idx * neurons + neuron_idx] += residual_inputs[batch_idx * neurons + neuron_idx];
	}
}

convolutional_layer::convolutional_layer() { 
	weights = nullptr;
	bias = nullptr;
	d_weights = nullptr;
	d_bias = nullptr;
	forward_output = nullptr;
	backward_input = nullptr;
	kernals = 0;
	kernal_size = 0;
	channels = 0;
	padding = 0;
	stride = 0;
	input_size = 0;
	output_size = 0;
	batch_size = 0;
	neurons = 0;
	inputs = 0;
}

convolutional_layer::convolutional_layer(size_t _input_size, size_t _channels, size_t _kernals, size_t _kernal_size, size_t _stride, size_t _padding, activation_functions _layer_activation_function) {

	kernals = _kernals;
	kernal_size = _kernal_size;
	channels = _channels;
	padding = _padding;
	stride = _stride;
	input_size = _input_size;
	batch_size = 0;
	
	output_size = (input_size + (2 * padding) - kernal_size) / stride + 1;
	neurons = output_size * output_size * kernals;
	inputs = input_size * input_size * channels;

	weights = (double*)malloc(kernal_size * kernal_size * channels * kernals * sizeof(double));
	d_weights = (double*)malloc(kernal_size * kernal_size * channels * kernals * sizeof(double));
	bias = (double*)malloc(kernals * sizeof(double));
	d_bias = (double*)malloc(kernals * sizeof(double));

	forward_output = nullptr;
	backward_input = nullptr;
	layer_activation_function = _layer_activation_function;

	if (weights == nullptr || d_weights == nullptr || bias == nullptr || d_bias == nullptr) {
		std::cerr << "Error: Could not allocate memory in convolutional layer" << std::endl;
		exit(EXIT_FAILURE);
	}  

	std::uniform_real_distribution<double> distribution(-1.0, 1.0);
	std::mt19937 generator;

	for (size_t i = 0; i < kernals; i++) {
		bias[i] = distribution(generator);
		for (size_t j = 0; j < channels * kernal_size * kernal_size; j++) {
			weights[i * kernal_size * kernal_size * channels + j] = distribution(generator);
		}
	}	
}

convolutional_layer::~convolutional_layer() {
	free(weights);
	free(d_weights);
	free(bias);
	free(d_bias);
	free(forward_output);
	free(backward_input);
}

void convolutional_layer::forward(const std::vector<std::vector<double>>& batched_inputs) {
	std::cerr << "Error: Auto de-flattening not supported at this time" << std::endl;
	exit(EXIT_FAILURE);
}

void convolutional_layer::forward(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_inputs) {
	
	double* input_arr = (double*)malloc(batched_inputs.size() * inputs * sizeof(double));
	
	if (input_arr == nullptr) {
		std::cerr << "Error: Could not allocate memory in convolutional_layer for forward pass" << std::endl;
		exit(EXIT_FAILURE);
	}

	for (size_t i = 0; i < batched_inputs.size(); i++) {
		if (batched_inputs[i].size() != channels) {
			std::cerr << "Error: Inputs incomptibale shape with convolutional layer" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (size_t j = 0; j < channels; j++) {

			if (batched_inputs[i][j].size() != input_size) {
				std::cerr << "Error: Inputs incomptibale shape with convolutional layer" << std::endl;
				exit(EXIT_FAILURE);
			}

			for (size_t z = 0; z < input_size; z++) {

				if (batched_inputs[i][j][z].size() != input_size) {
					std::cerr << "Error: Inputs incomptibale shape with convolutional layer" << std::endl;
					exit(EXIT_FAILURE);
				}

				memcpy(input_arr + z * input_size + j * input_size * input_size + i * channels * input_size * input_size, batched_inputs[i][j][z].data(), input_size * sizeof(double));
			}
		}
	}
	forward(input_arr, inputs, batched_inputs.size());

	free(input_arr);
}

void convolutional_layer::forward(double* batched_inputs, size_t _input_size, size_t _batch_size) {
	
	if (inputs != _input_size) {
		std::cerr << "Error: Incompatible input shape with convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (batch_size != _batch_size) {
		if (forward_output != nullptr) free(forward_output);
		forward_output = (double*)malloc(_batch_size * neurons * sizeof(double));
		
		if (backward_input != nullptr) {
			free(backward_input);
			backward_input = nullptr;
		}

		if (forward_output == nullptr) {
			std::cerr << "Error: Could not allocate memory in convolutional layer for forward pass" << std::endl;
			exit(EXIT_FAILURE);
		}

		batch_size = _batch_size;
	}

	if (backward_input != nullptr) {
		memset(backward_input, 0, batch_size * neurons * sizeof(double));
	}

	double* cuda_batched_inputs = nullptr;
	double* cuda_weights = nullptr;
	double* cuda_bias = nullptr;
	double* cuda_forward_output = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&cuda_batched_inputs, batch_size * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_weights, kernals * channels * kernal_size * kernal_size * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_bias, kernals * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_forward_output, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_batched_inputs, batched_inputs, batch_size * inputs * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_weights, weights, kernals * channels * kernal_size * kernal_size * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_bias, bias, kernals * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	dim3 blocks(batch_size/6 + 1, kernals/6 + 1, (output_size * output_size)/6 + 1);
	dim3 threads(6, 6, 6);
	Cuda_Convolutional_Layer_Forward_Pass<<<blocks, threads>>>(cuda_batched_inputs, cuda_weights, cuda_bias, cuda_forward_output, batch_size, kernals, kernal_size, padding, output_size, channels, input_size, stride);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: failed to launch convolutional layer forward pass kernal" << std::endl;
		exit(error_code);
	}

	if (layer_activation_function == activation_functions::Sigmoid) {
		dim3 blocks_2d(neurons/16 + 1, batch_size/16 + 1);
		dim3 threads_2d(16, 16);
		Cuda_Sigmoid_Activation_Forward_Pass<<<blocks_2d, threads_2d>>>(cuda_forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Rectified_Linear) {
		dim3 blocks_2d(neurons/16 + 1, batch_size/16 + 1);
		dim3 threads_2d(16, 16);
		Cuda_Rectified_Linear_Activation_Forward_Pass<<<blocks_2d, threads_2d>>>(cuda_forward_output, batch_size, neurons);
	}
	
	if (layer_activation_function != activation_functions::Linear && (error_code = hipGetLastError()) != hipError_t::hipSuccess) {
		std::cerr << "Error: failed to launch convolutional layer forward activation function kernal" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(forward_output, cuda_forward_output, batch_size * neurons * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to host failed" << std::endl;
		exit(error_code);
	}

	hipFree(cuda_batched_inputs);
	hipFree(cuda_weights);
	hipFree(cuda_bias);
	hipFree(cuda_forward_output);
}

void convolutional_layer::forward(double* batched_inputs, double* residual_inputs, size_t _input_size, size_t _batch_size) {

	if (_input_size != inputs) {
		std::cerr << "Error: Batched_inputs of invalid shape to connect to dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (_batch_size != batch_size) {
		free(forward_output);
		free(backward_input);
		backward_input = nullptr;
		forward_output = (double*)malloc(_batch_size * neurons * sizeof(double));
		if (forward_output == nullptr) {
			std::cerr << "Error: Could not allocate memory for forward pass in dense_layer" << std::endl;
			exit(EXIT_FAILURE);
		}
		batch_size = _batch_size;
	}

	if (backward_input != nullptr) {
		memset(backward_input, 0, batch_size * neurons * sizeof(double));
	}

	double* cuda_batched_inputs = nullptr;
	double* cuda_residual_inputs = nullptr;
	double* cuda_forward_output = nullptr;
	double* cuda_weights = nullptr;
	double* cuda_bias = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&cuda_batched_inputs, batch_size * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_forward_output, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)cuda_weights, kernals * channels * kernal_size * kernal_size * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_bias, kernals * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_batched_inputs, batched_inputs, batch_size * inputs * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_weights, weights, kernals * channels * kernal_size * kernal_size * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_bias, bias, kernals * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	dim3 blocks(batch_size/6 + 1, kernals/6 + 1, (output_size * output_size)/6 + 1);
	dim3 threads(6, 6, 6);

	Cuda_Convolutional_Layer_Forward_Pass<<<blocks, threads>>>(cuda_batched_inputs, cuda_weights, cuda_bias, cuda_forward_output, batch_size, kernals, kernal_size, padding, output_size, channels, input_size, stride);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch forward pass kernal" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)cuda_residual_inputs, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_residual_inputs, residual_inputs, batch_size * neurons * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	dim3 blocks_2d(neurons / 16 + 1, batch_size / 16 + 1);
	dim3 threads_2d(16, 16);

	Cuda_Matrix_Addition<<<blocks_2d, threads_2d>>>(cuda_residual_inputs, cuda_forward_output, batch_size, neurons);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch matrix addition kernal" << std::endl;
		exit(error_code);
	}

	if (layer_activation_function == activation_functions::Sigmoid) {
		Cuda_Sigmoid_Activation_Forward_Pass<<<blocks_2d, threads_2d>>>(cuda_forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Rectified_Linear) {
		Cuda_Rectified_Linear_Activation_Forward_Pass<<<blocks_2d, threads_2d>>>(cuda_forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Softmax) {
		//TODO: Implement softmax.
	}

	if (layer_activation_function != activation_functions::Linear && (error_code = hipGetLastError()) != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch forward activation function kernal" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: cudaDeviceSynchronice failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(forward_output, cuda_forward_output, batch_size * neurons * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to host failed" << std::endl;
		exit(error_code);
	}

	hipFree(cuda_forward_output);
	hipFree(cuda_batched_inputs);
	hipFree(cuda_bias);
	hipFree(cuda_weights);
	hipFree(cuda_residual_inputs);
}

void convolutional_layer::forward(const layer* prev_layer) {

	if (prev_layer->neurons != inputs) {
		std::cerr << "Error: Prev_layer of invalid input shape size to connect to convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	forward(prev_layer->forward_output, prev_layer->neurons, prev_layer->batch_size);
}

void convolutional_layer::forward(const layer* prev_layer, const layer* residual_layer) {

	if (prev_layer->neurons != inputs || residual_layer->neurons != neurons || prev_layer->batch_size != residual_layer->batch_size ) {
		std::cerr << "Error: Prev_layer or residual_layer of invalid input shape to connect to convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	forward(prev_layer->forward_output, residual_layer->forward_output, prev_layer->neurons, prev_layer->batch_size);
}


double convolutional_layer::loss(const std::vector<std::vector<double>>& batched_targets) const {

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Incompatible batch size, cannot calculate loss in convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	double result = 0.0;

	for (size_t i = 0; i < batch_size; i++) {
		
		if (batched_targets[i].size() != neurons) {
			std::cerr << "Error: Incompatible input shape, cannot calculate loss in convolutional_layer" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (size_t j = 0; j < neurons; j++) {
			result += ((forward_output[i * neurons + j] - batched_targets[i][j]) * (forward_output[i * neurons + j] - batched_targets[i][j])) / (double)(neurons * batch_size);
		}
	}
	
	return result;
}
double convolutional_layer::loss(const std::vector<int>& batched_targets) const {

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Incompatible batch size, cannot calculate loss in convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	//need to check my softmax math here.
	return 0.0;
}
double convolutional_layer::loss(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_targets) const {

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Incompatible batch size, cannot calculate loss in convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	double result = 0.0;
	
	for (size_t i = 0; i < batch_size; i++) {

		if (batched_targets[i].size() != kernals) {
			std::cerr << "Error: Incompatible input shape, cannot calculate loss in convolutional_layer" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (size_t j = 0; j < kernals; j++) {

			if (batched_targets[i][j].size() != output_size) {
				std::cerr << "Error: Incompatible input shape, cannot calculate loss in convolutional_layer" << std::endl;
				exit(EXIT_FAILURE);
			}

			for (size_t y = 0; y < output_size; y++) {

				if (batched_targets[i][j][y].size() != output_size) {
					std::cerr << "Error: Incompatible input shape, cannot calculate loss in convolutional_layer" << std::endl;
					exit(EXIT_FAILURE);
				}
				
				for (size_t x = 0; x < output_size; x++) {
					result += ((forward_output[i * neurons + j * output_size * output_size + y * output_size + x] - batched_targets[i][j][y][x]) * (forward_output[i * neurons + j * output_size * output_size + y * output_size + x] - batched_targets[i][j][y][x])) / (double)(neurons * batch_size);	
				}
			}
		}
	}

	return result;
}

void convolutional_layer::init_back_propigation(const std::vector<std::vector<double>>& batched_targets) {

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* input_arr = (double*)malloc(batch_size * neurons * sizeof(double));

	if (input_arr == nullptr) {
		std::cerr << "Error: Could not allocted memory in convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	for (size_t i = 0; i < batch_size; i++) {
		
		if (batched_targets[i].size() != neurons) {
			std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
			exit(EXIT_FAILURE);
		}

		memcpy(input_arr + i * neurons, batched_targets[i].data(), neurons * sizeof(double));
	}

	init_back_propigation(input_arr, neurons, batch_size);
}
void convolutional_layer::init_back_propigation(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_targets) {

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* input_arr = (double*)malloc(batch_size * neurons * sizeof(double));

	if (input_arr == nullptr) {
		std::cerr << "Error: Could not allocted memory in convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}


	for (size_t i = 0; i < batch_size; i++) {

		if (batched_targets[i].size() != kernals) {
			std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (size_t j = 0; j < kernals; j++) {

			if (batched_targets[i][j].size() != output_size) {
				std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
				exit(EXIT_FAILURE);
			}

			for (size_t y = 0; y < output_size; y++) {
				
				if (batched_targets[i][j][y].size() != output_size) {
					std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
					exit(EXIT_FAILURE);
				}

				memcpy(input_arr + i * neurons + j * output_size * output_size + y * output_size, batched_targets[i][j][y].data(), output_size * sizeof(double));
			}
		}
	}

	init_back_propigation(input_arr, neurons, batch_size);
}
void convolutional_layer::init_back_propigation(double* batched_targets, size_t _input_size, size_t _batch_size) {

	if (_input_size != neurons || _batch_size != batch_size) {
		std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (backward_input == nullptr) {
		backward_input = (double*)malloc(batch_size * neurons * sizeof(double));
		if (backward_input == nullptr) {
			std::cerr << "Error: Unable to allocate memory for backpropigation in convolutional_layer" << std::endl;
			exit(EXIT_FAILURE);
		}
	}

	double* cuda_forward_output = nullptr;
	double* cuda_batched_targets = nullptr;
	double* cuda_backward_input = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&cuda_forward_output, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_batched_targets, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_backward_input, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_forward_output, forward_output, batch_size * neurons * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_batched_targets, batched_targets, batch_size * neurons * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	dim3 blocks(neurons / 16 + 1, batch_size / 16 + 1);
	dim3 threads(16, 16);
	
	Cuda_Convolutional_Layer_Init_Back_Propigation<<<blocks, threads>>>(cuda_batched_targets, cuda_forward_output, cuda_backward_input, batch_size, neurons);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch init back propigation kernal in convolutional_layer" << std::endl;
		exit(error_code);
	}

	if (layer_activation_function == activation_functions::Sigmoid) {
		Cuda_Sigmoid_Activation_Backward_Pass<<<blocks, threads>>>(cuda_backward_input, cuda_forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Rectified_Linear) {
		Cuda_Rectified_Linear_Activation_Backward_Pass<<<blocks, threads>>>(backward_input, forward_output, batch_size, neurons);
	}

	if (layer_activation_function != activation_functions::Linear && (error_code = hipGetLastError()) != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch backward activation function kernal" << std::endl;
		exit(error_code);
	}
	
	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}
	
	error_code = hipMemcpy(backward_input, cuda_backward_input, batch_size * neurons * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to host failed" << std::endl;
		exit(error_code);
	}

	hipFree(cuda_forward_output);
	hipFree(cuda_batched_targets);
	hipFree(cuda_backward_input);
}

void convolutional_layer::backward(const std::vector<std::vector<double>>& batched_inputs) {
	
	if (batched_inputs.size() != batch_size) {
		std::cerr << "Error: Batched_inputs of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* input_arr = (double*)malloc(batch_size * inputs * sizeof(double));

	if (input_arr == nullptr) {
		std::cerr << "Error: Could not allocate memory in convolutional layer for backpropigation" << std::endl;
		exit(EXIT_FAILURE);
	}

	for (size_t i = 0; i < batch_size; i++) {

		if (batched_inputs[i].size() != inputs) {
			std::cerr << "Error: Batched_inputs of incompatible input shape" << std::endl;
			exit(EXIT_FAILURE);
		}

		memcpy(input_arr + i * inputs, batched_inputs[i].data(), inputs * sizeof(double));
	}

	backward(input_arr, inputs, batch_size);

	free(input_arr);
}

void convolutional_layer::backward(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_inputs) {

	if (batched_inputs.size() != batch_size) {
		std::cerr << "Error: Batched_inputs of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* input_arr = (double*)malloc(batch_size * inputs * sizeof(double));

	if (input_arr == nullptr) {
		std::cerr << "Error: Could not allocate memory in convolutional layer for backpropigation" << std::endl;
		exit(EXIT_FAILURE);
	}

	for (size_t i = 0; i < batch_size; i++) {
		
		if (batched_inputs[i].size() != channels) {
			std::cerr << "Error: Batched_inputs of incompatible input shape" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (size_t j = 0; j < channels; j++) {

			if (batched_inputs[i][j].size() != input_size) {
				std::cerr << "Error: Batched_inputs of incompatible input shape" << std::endl;
				exit(EXIT_FAILURE);
			}

			for (size_t y = 0; y < input_size; y++) {

				if (batched_inputs[i][j][y].size() != input_size) {
					std::cerr << "Error: Batched_inputs of incompatible input shape" << std::endl;
					exit(EXIT_FAILURE);
				}

				memcpy(input_arr + i * inputs + j * input_size * input_size + y * input_size, batched_inputs[i][j][y].data(), input_size * sizeof(double));
			}
		}
	}
	  
	backward(input_arr, inputs, batch_size);
	free(input_arr);
}
void convolutional_layer::backward(double* batched_inputs, size_t _input_size, size_t _batch_size) {

	if (backward_input == nullptr) {
		std::cerr << "Error: Convolutional_layer not initialized for backward pass" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (inputs != _input_size || batch_size != _batch_size) {
		std::cerr << "Error: Batched_inputs of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* cuda_d_weights = nullptr;
	double* cuda_d_bias = nullptr;
	double* cuda_batched_input = nullptr;
	double* cuda_backward_input = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&cuda_d_weights, kernals * channels * kernal_size * kernal_size * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_d_bias, kernals * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_batched_input, batch_size * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_backward_input, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_batched_input, batched_inputs, batch_size * inputs * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_backward_input, backward_input, batch_size * neurons * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	dim3 blocks(kernals/6 + 1, channels/6 + 1, (kernal_size * kernal_size)/6 + 1);
	dim3 threads(6, 6, 6);

	Cuda_Convolutional_Layer_First_Backward_Pass<<<blocks, threads>>>(cuda_batched_input, cuda_backward_input, cuda_d_weights, batch_size, kernals, channels, kernal_size, input_size, output_size, stride, padding);
	
	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch kernal for backward pass in convolutional_layer" << std::endl;
		exit(error_code);
	}

	Cuda_Convolutional_Layer_Second_Backward_Pass <<<kernals / 16 + 1, 16 >> > (cuda_backward_input, cuda_d_bias, batch_size, kernals, output_size);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch kernal for backward pass in convolutional_layer" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(d_weights, cuda_d_weights, kernals * channels * kernal_size * kernal_size * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to host failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(d_bias, cuda_d_bias, kernals * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to host failed" << std::endl;
		exit(error_code);
	}
	
	hipFree(cuda_d_weights);
	hipFree(cuda_d_bias);
	hipFree(cuda_backward_input);
	hipFree(cuda_batched_input);
}
void convolutional_layer::backward(layer* prev_layer) {

	if (prev_layer->batch_size != batch_size || prev_layer->neurons != inputs) {
		std::cerr << "Error: Prev_layer of invalid input shape or batch size to connect to convolutional_layer" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	backward(prev_layer->forward_output, prev_layer->neurons, prev_layer->batch_size);

	if (prev_layer->backward_input == nullptr) {

		prev_layer->backward_input = (double*)malloc(batch_size * inputs * sizeof(double));
		if (prev_layer->backward_input == nullptr) {
			std::cerr << "Error: Could not allocate memory for backpropigation" << std::endl;
			exit(EXIT_FAILURE);
		}
	}

	double* cuda_weights = nullptr;
	double* cuda_backward_input = nullptr;
	double* cuda_prev_layer_backward_input = nullptr;
	double* cuda_prev_layer_forward_output = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&cuda_weights, kernals * channels * kernal_size * kernal_size * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_backward_input, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_prev_layer_backward_input, batch_size * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_weights, weights, kernals * channels * kernal_size * kernal_size * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_backward_input, backward_input, batch_size * neurons * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}
	
	dim3 blocks(batch_size/6 + 1, channels/6 + 1, (input_size * input_size)/6 + 1);
	dim3 threads(6, 6, 6);
	
	Cuda_Convolution_Layer_Partial_Derivitive_of_Loss<<<blocks, threads>>>(cuda_backward_input, cuda_weights, cuda_prev_layer_backward_input, batch_size, kernals, channels, kernal_size, input_size, padding, stride, output_size);
	
	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch partial derivitive of loss kernal" << std::endl;
		exit(error_code);
	}
	
	if (prev_layer->layer_activation_function != activation_functions::Linear) {

		error_code = hipMalloc((void**)&cuda_prev_layer_forward_output, batch_size * inputs * sizeof(double));
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMalloc failed" << std::endl;
			exit(error_code);
		}

		error_code = hipMemcpy(cuda_prev_layer_forward_output, prev_layer->forward_output, batch_size * inputs * sizeof(double), hipMemcpyHostToDevice);
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMemcpy to device failed" << std::endl;
			exit(error_code);
		}

		dim3 blocks_2d(batch_size/16 + 1, inputs/16 + 1);
		dim3 threads_2d(16, 16);

		if (layer_activation_function == activation_functions::Sigmoid) {
			Cuda_Sigmoid_Activation_Backward_Pass<<<blocks_2d, threads_2d>>>(cuda_prev_layer_backward_input, cuda_prev_layer_forward_output, batch_size, inputs);
		}
		else if (layer_activation_function == activation_functions::Rectified_Linear) {
			Cuda_Rectified_Linear_Activation_Backward_Pass<<<blocks_2d, threads_2d>>>(cuda_prev_layer_backward_input, cuda_prev_layer_forward_output, batch_size, inputs);
		}

		error_code = hipGetLastError();
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: Failed to launch backward activation function kernal" << std::endl;
			exit(error_code);
		}
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}
	
	error_code = hipMemcpy(prev_layer->backward_input, cuda_prev_layer_backward_input, batch_size * inputs * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to host failed" << std::endl;
		exit(error_code);
	}

	hipFree(cuda_weights);
	hipFree(cuda_backward_input);
	hipFree(cuda_prev_layer_backward_input);
	if (cuda_prev_layer_forward_output != nullptr) hipFree(cuda_prev_layer_forward_output);
}

void convolutional_layer::update_paramters(double learning_rate) {

	for (int i = 0; i < kernals * channels * kernal_size * kernal_size; i++) {
		weights[i] -= d_weights[i] * learning_rate;
	}
	
	for (int i = 0; i < kernals; i++) {
		bias[i] -= d_bias[i] * learning_rate;
	}
}