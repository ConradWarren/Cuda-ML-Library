#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "layer.hpp"
#include <iostream>
#include <stdio.h>

__global__ void Cuda_Dense_Layer_Forward_Pass(double* batched_inputs, double* weights, double* bias, double* forward_output, size_t inputs, size_t neurons, size_t batch_size) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		
		forward_output[batch_idx * neurons + neuron_idx] = bias[neuron_idx];
		for (size_t i = 0; i < inputs; i++) {
			forward_output[batch_idx*neurons + neuron_idx] += weights[neuron_idx*neurons + i] * batched_inputs[batch_idx * neurons + i];
		}
	}
}

__global__ void Cuda_Init_Back_Propigation(double* batched_targets, double* forward_ouput, double* backward_input, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		backward_input[batch_idx * neurons + neuron_idx] = 2.0 * (forward_ouput[batch_idx * neurons + neuron_idx]) / (batch_size * neurons);
	}
}

__global__ void Cuda_Sigmoid_Activation_Forward_Pass(double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		forward_output[batch_idx * neurons + neuron_idx] = 1.0 / (1.0 + std::powf(2.71828182846, -forward_output[batch_idx * neurons + neurons]));
	}
}

__global__ void Cuda_Rectified_Linear_Activation_Forward_Pass(double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons && forward_output[batch_idx * neurons + neuron_idx] < 0.0) {
		forward_output[batch_idx * neurons + neuron_idx] = 0.0;
	}
}

dense_layer::dense_layer() {
	neurons = 0;
	inputs = 0;
	weights = nullptr;
	bias = nullptr;
	forward_output = nullptr;
	layer_activation_function = activation_functions::Linear;
}

dense_layer::dense_layer(size_t _inputs, size_t _neurons) {

	neurons = _neurons;
	inputs = _inputs;
	batch_size = 0;
	layer_activation_function = activation_functions::Sigmoid;

	weights = (double*)malloc(inputs * neurons * sizeof(double));
	bias = (double*)malloc(neurons * sizeof(double));
	d_weights = (double*)malloc(inputs * neurons * sizeof(double));
	d_bias = (double*)malloc(neurons * sizeof(double));

	forward_output = nullptr;
	backward_input = nullptr;

	if (bias == nullptr || weights == nullptr) {
		std::cerr << "Error: Could not allocate memory for layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	for (size_t i = 0; i < neurons; i++) {
		bias[i] = (double)i;
		for (size_t j = 0; j < inputs; j++) {
			weights[i*inputs + j] = (double)(i * inputs + j);
		}
	}

}

dense_layer::~dense_layer() {
	free(weights);
	free(bias);
	free(forward_output);
	free(backward_input);
}

void dense_layer::forward(const std::vector<std::vector<double>>& batched_inputs) {

	double* input_arr = (double*)malloc(batched_inputs.size() * inputs * sizeof(double));

	if (input_arr == nullptr) {
		std::cerr << "Error: Could not allocate memory in dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < batched_inputs.size(); i++) {

		if (batched_inputs[i].size() != inputs) {
			std::cerr << "Error: batched_inputs of invalid shape" << std::endl;
			exit(EXIT_FAILURE);
		}
		memcpy(input_arr + (inputs * i), batched_inputs[i].data(), inputs * sizeof(double));
	}

	forward(input_arr, inputs, batched_inputs.size());

	free(input_arr);
}
void dense_layer::forward(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_inputs) {
	std::cerr << "Error: auto flattening not currently supported" << std::endl;
	exit(EXIT_FAILURE);
}

void dense_layer::forward(double* batched_inputs, size_t _input_size, size_t _batch_size) {

	if (_input_size != inputs) {
		std::cerr << "Error: Incompatible input for dense layer of shape " << inputs << " " << neurons << std::endl;
		exit(EXIT_FAILURE);
	}

	if (_batch_size != batch_size || forward_output == nullptr) {

		if (forward_output != nullptr) free(forward_output);
		if (backward_input != nullptr) {
			free(backward_input);
			backward_input = nullptr;
		}
		forward_output = (double*)malloc(neurons * _batch_size * sizeof(double));

		if (forward_output == nullptr) {
			std::cerr << "Error: Could not allocate memory for dense_layer output" << std::endl;
			exit(EXIT_FAILURE);
		}
		batch_size = _batch_size;
	}

	double* cuda_batched_inputs = nullptr;
	double* cuda_weights = nullptr;
	double* cuda_bias = nullptr;
	double* cuda_forward_output = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&cuda_batched_inputs, batch_size * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_weights, neurons * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_bias, neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_forward_output, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_batched_inputs, batched_inputs, batch_size * inputs * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_weights, weights, neurons * inputs * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_bias, bias, neurons * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}
	
	int block_size = (batch_size > neurons) ? ((int)batch_size + 16 - 1) / 16 : ((int)neurons + 16 - 1) / 16;
	dim3 blocks(block_size, block_size);
	dim3 threads(16, 16);

	Cuda_Dense_Layer_Forward_Pass<<<blocks, threads>>>(cuda_batched_inputs, cuda_weights, cuda_bias, cuda_forward_output, inputs, neurons, batch_size);
	
	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Forward Pass Kernal Launch failed" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}

	if (layer_activation_function == activation_functions::Sigmoid) {
		Cuda_Sigmoid_Activation_Forward_Pass<<<blocks, threads>>>(cuda_forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Rectified_Linear) {
		Cuda_Rectified_Linear_Activation_Forward_Pass<<<blocks, threads>>>(cuda_forward_output, batch_size, neurons);
	}

	if (layer_activation_function != activation_functions::Linear && (error_code = hipGetLastError()) != hipError_t::hipSuccess) {
		std::cerr << "Error: Activation function kernal launch failed" << std::endl;
		exit(error_code);
	}

	if (layer_activation_function != activation_functions::Linear && (error_code = hipDeviceSynchronize()) != hipError_t::hipSuccess) {
		std::cerr << "Error : cudaDeciveSynchronize failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(forward_output, cuda_forward_output, batch_size * neurons * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to host failed" << std::endl;
		exit(error_code);
	}

	hipFree(cuda_batched_inputs);
	hipFree(cuda_weights);
	hipFree(cuda_bias);
	hipFree(cuda_forward_output);
}

void dense_layer::forward(const layer* prev_layer) {
	forward(prev_layer->forward_output, prev_layer->neurons, prev_layer->batch_size);
}

double dense_layer::loss(std::vector<std::vector<double>>& batched_targets) const {

	if (forward_output == nullptr) {
		std::cerr << "Error: No forwad output in dense layer to calculate loss" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Incompatible batch size, cannot calculate loss" << std::endl;
		exit(EXIT_FAILURE);
	}

	double result = 0.0;

	for (int i = 0; i < batch_size; i++) {

		if (batched_targets[i].size() != neurons) {
			std::cerr << "Error: Invalid input shape" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (int j = 0; j < neurons; j++) {
			result += ((forward_output[i * neurons + j] - batched_targets[i][j]) * (forward_output[i * neurons + j] - batched_targets[i][j])) / (double)(batch_size * neurons);
		}
	}

	return result;
}

double dense_layer::loss(std::vector<int>& batched_targets) const {

	//TODO: activation flag check

	if (forward_output == nullptr) {
		std::cerr << "Error: No forwad output in dense layer to calculate loss" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Incompatible batch size, cannot calculate loss" << std::endl;
		exit(EXIT_FAILURE);
	}
	//TODO: need to check math here again.

	return 0;
}
double dense_layer::loss(std::vector<std::vector<std::vector<std::vector<double>>>>& batched_targets) const {
	std::cerr << "Error: auto flattening not currently supported" << std::endl;
	exit(EXIT_FAILURE);
	return 0;
}

void dense_layer::init_back_propigation(std::vector<std::vector<double>>& batched_targets) {
	
	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Incompatible batch size, cannot calculate loss" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* input_arr = (double*)malloc(batch_size * neurons * sizeof(double));

	if (input_arr == nullptr) {
		std::cerr << "Error: Could not allocated memory in dense layer for backpropigation" << std::endl;
		exit(EXIT_FAILURE);
	}

	for (size_t i = 0; i < batch_size; i++) {
		
		if (batched_targets[i].size() != neurons) {
			std::cerr << "Error: batched targets of invalid shape" << std::endl;
			exit(EXIT_FAILURE);
		}
		memcpy(input_arr + i * neurons, batched_targets[i].data(), neurons * sizeof(double));
	}

	init_back_propigation(input_arr, neurons, batch_size);

	free(input_arr);
}

void dense_layer::init_back_propigation(std::vector<std::vector<std::vector<std::vector<double>>>>& batched_targets) {
	std::cerr << "Error: auto flattening not currently supported" << std::endl;
	exit(EXIT_FAILURE);
}

void dense_layer::init_back_propigation(double* batched_targets, size_t input_size, size_t _batch_size) {
	
	//probally not worth it too load this into a kenral but were going to do it anyway.

	if (batch_size != _batch_size || input_size != neurons) {
		std::cerr << "Error: Invalid input size for dense layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (backward_input == nullptr) {
		backward_input = (double*)malloc(batch_size * neurons * sizeof(double));
		if (backward_input == nullptr) {
			std::cerr << "Error: Could not allocate memory in dense layer for backpropigation" << std::endl;
			exit(EXIT_FAILURE);
		}
	}

	double* cuda_backward_input = nullptr;
	double* cuda_batched_targets = nullptr;
	double* cuda_forward_output = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&cuda_backward_input, neurons * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_batched_targets, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&cuda_forward_output, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_batched_targets, batched_targets, batch_size * neurons * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to Device failed" << std::endl;
		exit(error_code);
	}
	error_code = hipMemcpy(cuda_forward_output, forward_output, batch_size * neurons * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to Device failed" << std::endl;
		exit(error_code);
	}

	//kernal call goes here.
	int block_size = (batch_size > neurons) ? ((int)batch_size + 16 - 1) / 16 : ((int)neurons + 16 - 1) / 16;
	dim3 blocks(block_size, block_size);
	dim3 threads(16, 16);

	Cuda_Init_Back_Propigation<<<blocks, threads>>>(cuda_batched_targets, cuda_forward_output, cuda_backward_input, batch_size, neurons);
	
	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch kernal" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(backward_input, cuda_backward_input, batch_size * neurons * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: cudaMemcy to Host failed" << std::endl;
	}
	
	hipFree(cuda_backward_input);
	hipFree(cuda_batched_targets);
	hipFree(cuda_forward_output);
}