#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "layer.hpp"
#include <iostream>
#include <stdio.h>
#include <random>

__global__ static void Cuda_Dense_Layer_Forward_Pass(double* batched_inputs, double* weights, double* bias, double* forward_output, size_t inputs, size_t neurons, size_t batch_size) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		
		forward_output[batch_idx * neurons + neuron_idx] = bias[neuron_idx];
		for (size_t i = 0; i < inputs; i++) {
			forward_output[batch_idx*neurons + neuron_idx] += weights[neuron_idx*inputs + i] * batched_inputs[batch_idx * inputs + i];
		}
	}
}

__global__ static void Cuda_Dense_Layer_Init_Back_Propigation(double* batched_targets, double* forward_ouput, double* backward_input, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		backward_input[batch_idx * neurons + neuron_idx] = 2.0 * (forward_ouput[batch_idx * neurons + neuron_idx] - batched_targets[batch_idx * neurons + neuron_idx]) / (double)(batch_size * neurons);
	}
}

__global__ static void Cuda_Dense_Layer_Init_Cross_Catigorial_Loss_Back_Propigation(unsigned int* batched_targets, double* forward_output, double* backward_input, size_t batch_size, size_t neurons) {
	
	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		backward_input[batch_idx * neurons + neuron_idx] = forward_output[batch_idx * neurons + neuron_idx]/(double)batch_size;
		if (neuron_idx == batched_targets[batch_idx]) backward_input[batch_idx * neurons + neuron_idx] -= 1.0 / (double)batch_size;
	}
}

__global__ static void Cuda_Dense_Layer_First_Backward_Pass(double* batched_inputs, double* backward_input, double* d_weights, size_t batch_size, size_t neurons, size_t inputs){
	
	size_t neuron_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t input_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (neuron_idx < neurons && input_idx < inputs) {

		d_weights[neuron_idx * inputs + input_idx] = 0.0;
		for (size_t i = 0; i < batch_size; i++) {
			d_weights[neuron_idx * inputs + input_idx] += batched_inputs[i * inputs + input_idx] * backward_input[i * neurons + neuron_idx];
		}
	}
}

__global__ static void Cuda_Dense_Layer_Second_Backward_Pass(double* backward_input, double* d_bias, size_t batch_size, size_t neurons) {

	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (neuron_idx < neurons) {
		d_bias[neuron_idx] = 0.0;
		for (size_t i = 0; i < batch_size; i++) {
			d_bias[neuron_idx] += backward_input[(i * neurons) + neuron_idx];
		}
	}
}

__global__ static void Cuda_Sigmoid_Activation_Forward_Pass(double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		forward_output[batch_idx * neurons + neuron_idx] = 1.0 / (1.0 + std::powf(2.71828182846, -forward_output[batch_idx * neurons + neuron_idx]));
	}
}

__global__ static void Cuda_Sigmoid_Activation_Backward_Pass(double* backward_input, double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		backward_input[batch_idx * neurons + neuron_idx] *= forward_output[batch_idx * neurons + neuron_idx] * (1.0 - forward_output[batch_idx * neurons + neuron_idx]);
	}
}

__global__ static void Cuda_Rectified_Linear_Activation_Forward_Pass(double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons && forward_output[batch_idx * neurons + neuron_idx] < 0.0) {
		forward_output[batch_idx * neurons + neuron_idx] = 0.0;
	}
}

__global__ static void Cuda_Rectified_Linear_Activation_Backward_Pass(double* backward_input, double* forward_input, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons && forward_input[batch_idx * neurons + neuron_idx] == 0.0) {
		backward_input[batch_idx * neurons + neuron_idx] = 0.0;
	}
}

__global__ static void Cuda_Softmax_Activation_Forward_Pass(double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		forward_output[batch_idx * neurons + neuron_idx] = std::powf(2.71828182846, forward_output[batch_idx * neurons + neuron_idx]);
	}

	__syncthreads();

	double sum = 0;

	if (batch_idx < batch_size && neuron_idx < neurons) {
		for (int i = 0; i < neurons; i++) {
			sum += forward_output[batch_idx * neurons + i];
		}
	}
	
	__syncthreads();

	if (batch_idx < batch_size && neuron_idx < neurons) {
		forward_output[batch_idx * neurons + neuron_idx] /= sum;
	}
}

__global__ static void Cuda_Softmax_Activation_Bakcward_Pass(double* forward_output, double* backward_input, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	double sum = 0.0;
	if (batch_idx < batch_size && neuron_idx < neurons) {

		for (int i = 0; i < neurons; i++) {
			sum += backward_input[batch_idx * neurons + i] * forward_output[batch_idx * neurons + i];
		}
	}

	__syncthreads();

	if (batch_idx < batch_size && neuron_idx < neurons) {
		backward_input[batch_idx * neurons + neuron_idx] = (backward_input[batch_idx * neurons + neuron_idx] - sum) * forward_output[batch_idx * neurons + neuron_idx];
	}
}

__global__ static void Cuda_Partial_Derivitive_of_Loss(double* backward_input, double* weights, double* prev_layer_backward_input, size_t batch_size, size_t inputs, size_t neurons) {
	
	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t input_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (batch_idx < batch_size && input_idx < inputs) {
		for (int i = 0; i < neurons; i++) {
			prev_layer_backward_input[batch_idx * inputs + input_idx] += backward_input[batch_idx * neurons + i] * weights[i * inputs + input_idx];
		}
	}
}

__global__ static void Cuda_Matix_Addition(double* residual_batched_inputs, double* forward_output, size_t batch_size, size_t neurons) {

	size_t batch_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t neuron_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if (batch_idx < batch_size && neuron_idx < neurons) {
		forward_output[batch_idx * neurons + neuron_idx] += residual_batched_inputs[batch_idx * neurons + neuron_idx];
	}
}

__global__ static void Cuda_Graident_Decent(double* d_weights, double* d_bias, double* weights, double* bias,double learning_rate, size_t neurons, size_t inputs) {

	size_t neuron_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
	size_t input_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (neuron_idx < neurons && input_idx < inputs) {
		weights[neuron_idx * inputs + input_idx] -= d_weights[neuron_idx * inputs + input_idx] * learning_rate;
		if (input_idx == 0) {
			bias[neuron_idx] -= d_bias[neuron_idx] * learning_rate;
		}
	}
	
}

dense_layer::dense_layer() {
	neurons = 0;
	inputs = 0;
	weights = nullptr;
	bias = nullptr;
	forward_output = nullptr;
	backward_input = nullptr;
	d_weights = nullptr;
	d_bias = nullptr;
	layer_activation_function = activation_functions::Linear;
}

dense_layer::dense_layer(size_t _inputs, size_t _neurons, activation_functions _layer_activation_function) {

	neurons = _neurons;
	inputs = _inputs;
	batch_size = 0;
	layer_activation_function = _layer_activation_function;
	hipError_t error_code;

	error_code = hipMalloc((void**)&d_weights, neurons * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
		exit(error_code);
	}

	error_code = hipMemset(d_weights, 0, neurons * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemset failed in dnese_layer" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&d_bias, neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
		exit(error_code);
	}

	error_code = hipMemset(d_bias, 0, neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemset failed in dense_layer" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&weights, neurons * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
		exit(error_code);
	}

	error_code = hipMalloc((void**)&bias, neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
		exit(error_code);
	}

	forward_output = nullptr;
	backward_input = nullptr;

	double* temp_weights = (double*)malloc(neurons * inputs * sizeof(double));
	double* temp_bias = (double*)malloc(neurons * sizeof(double));

	if (temp_weights == nullptr || temp_bias == nullptr) {
		std::cerr << "Error: Failed to allocate memory in dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	std::uniform_real_distribution<double> distribution(-1.0, 1.0);
	std::mt19937 generator;

	for (size_t i = 0; i < neurons; i++) {
		temp_bias[i] = distribution(generator);
		for (size_t j = 0; j < inputs; j++) {
			temp_weights[i * inputs + j] = distribution(generator);
		}
	}

	error_code = hipMemcpy(weights, temp_weights, neurons * inputs * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed in dense_layer" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(bias, temp_bias, neurons * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed in dense_layer" << std::endl;
		exit(error_code);
	}

	free(temp_weights);
	free(temp_bias);

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed in dense_layer" << std::endl;
		exit(error_code);
	}
}

dense_layer::~dense_layer() {
	hipFree(weights);
	hipFree(d_weights);
	hipFree(bias);
	hipFree(d_bias);
	hipFree(forward_output);
	hipFree(backward_input);
}

void dense_layer::forward(const std::vector<std::vector<double>>& batched_inputs) {

	double* input_arr = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&input_arr, batched_inputs.size() * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
		exit(error_code);
	}
	
	for (size_t i = 0; i < batched_inputs.size(); i++) {

		if (batched_inputs[i].size() != inputs) {
			std::cerr << "Error: batched_inputs of invalid shape" << std::endl;
			exit(EXIT_FAILURE);
		}
		error_code = hipMemcpy(input_arr + i * inputs, batched_inputs[i].data(), inputs * sizeof(double), hipMemcpyHostToDevice);
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMemcpy to device failed in dense_layer" << std::endl;
			exit(error_code);
		}
	}

	forward(input_arr, inputs, batched_inputs.size());
	hipFree(input_arr);
}
void dense_layer::forward(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_inputs) {
	
	double* input_arr = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&input_arr, batched_inputs.size() * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
		exit(error_code);
	}

	unsigned int current_size = 0;

	for (int i = 0; i < batched_inputs.size(); i++) {
		for (int j = 0; j < batched_inputs[j].size(); j++) {
			for (int y = 0; y < batched_inputs[i][j].size(); y++) {

				if (current_size + batched_inputs[i][j][y].size() > inputs) {
					std::cerr << "Error: Batched_inputs of incompatible input shape" << std::endl;
					exit(EXIT_FAILURE);
				}

				error_code = hipMemcpy(input_arr + i * inputs + current_size, batched_inputs[i][j][y].data(), batched_inputs[i][j][y].size() * sizeof(double), hipMemcpyHostToDevice);
				if (error_code != hipError_t::hipSuccess) {
					std::cerr << "Error: hipMemcpy to device failed in dense_layer" << std::endl;
					exit(error_code);
				}
				
				current_size += batched_inputs[i][j][y].size();
			}
		}

		if (current_size != inputs) {
			std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
			exit(EXIT_FAILURE);
		}
		current_size = 0;
	}
	
	forward(input_arr, inputs, batch_size);
	hipFree(input_arr);
}

void dense_layer::forward(double* batched_inputs, size_t _input_size, size_t _batch_size) {

	if (_input_size != inputs) {
		std::cerr << "Error: Incompatible input for dense layer of shape " << inputs << " " << neurons << std::endl;
		exit(EXIT_FAILURE);
	}

	hipError_t error_code;

	if (_batch_size != batch_size || forward_output == nullptr) {

		hipFree(forward_output);
		hipFree(backward_input);
		backward_input = nullptr;

		error_code = hipMalloc((void**)&forward_output, neurons * _batch_size * sizeof(double));
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
			exit(error_code);
		}

		batch_size = _batch_size;
	}
	
	dim3 blocks(neurons / 16 + 1, batch_size / 16 + 1);
	dim3 threads(16, 16);

	if (backward_input != nullptr) {
		error_code = hipMemset(backward_input, 0, batch_size * neurons * sizeof(double));
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMemset failed in dense_layer" << std::endl;
			exit(error_code);
		}
	}

	Cuda_Dense_Layer_Forward_Pass<<<blocks, threads>>>(batched_inputs, weights, bias, forward_output, inputs, neurons, batch_size);
	
	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch forward pass kernal in dense_layer" << std::endl;
		exit(error_code);
	}

	if (layer_activation_function == activation_functions::Sigmoid) {
		Cuda_Sigmoid_Activation_Forward_Pass<<<blocks, threads>>>(forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Rectified_Linear) {
		Cuda_Rectified_Linear_Activation_Forward_Pass<<<blocks, threads>>>(forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Softmax) {
		Cuda_Softmax_Activation_Forward_Pass<<<blocks, threads>>>(forward_output, batch_size, neurons);
	}

	if (layer_activation_function != activation_functions::Linear && (error_code = hipGetLastError()) != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch forward activation kernal in dense_layer" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}
}

void dense_layer::forward(double* batched_inputs, double* residual_batched_inputs, size_t _input_size, size_t _batch_size) {

	if (_input_size != inputs) {
		std::cerr << "Error: Batched_inputs of invalid shape to connect to dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	hipError_t error_code;

	if (batch_size != _batch_size || forward_output == nullptr) {
		hipFree(forward_output);
		hipFree(backward_input);
		backward_input = nullptr;
		
		error_code = hipMalloc((void**)&forward_output, _batch_size * neurons * sizeof(double));
		
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
			exit(error_code);
		}

		batch_size = _batch_size;
	}

	dim3 blocks(neurons / 16 + 1, batch_size / 16 + 1);
	dim3 threads(16, 16);

	if (backward_input != nullptr) {
		error_code = hipMemset(backward_input, 0, batch_size * neurons * sizeof(double));
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMemset failed in dense_layer" << std::endl;
			exit(error_code);
		}
	}

	Cuda_Dense_Layer_Forward_Pass<<<blocks, threads>>>(batched_inputs, weights, bias, forward_output, inputs, neurons, batch_size);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch forward pass kernal" << std::endl;
		exit(error_code);
	}

	Cuda_Matix_Addition<<<blocks, threads>>>(residual_batched_inputs, forward_output, batch_size, neurons);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch matrix addition kernal" << std::endl;
		exit(error_code);
	}
	
	if (layer_activation_function == activation_functions::Sigmoid) {
		Cuda_Sigmoid_Activation_Forward_Pass<<<blocks, threads>>>(forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Rectified_Linear) {
		Cuda_Rectified_Linear_Activation_Forward_Pass<<<blocks, threads>>>(forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Softmax) {
		Cuda_Softmax_Activation_Forward_Pass<<<blocks, threads>>>(forward_output, batch_size, neurons);
	}

	if (layer_activation_function != activation_functions::Linear && (error_code = hipGetLastError()) != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch activation function forward pass kernal" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}
}

void dense_layer::forward(const layer* prev_layer) {

	if (prev_layer->neurons != inputs) {
		std::cerr << "Error: Prev_layer of invalid input shape to connect to dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}
	forward(prev_layer->forward_output, prev_layer->neurons, prev_layer->batch_size);
}

void dense_layer::forward(const layer* prev_layer, const layer* residual_layer) {

	if (prev_layer->neurons != inputs || residual_layer->neurons != neurons || prev_layer->batch_size != residual_layer->batch_size) {
		std::cerr << "Error: Prev_layer_1 and prev_layer_2 of invalid input shape to connect to dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}
	forward(prev_layer->forward_output, residual_layer->forward_output, prev_layer->neurons, prev_layer->batch_size);
}

double dense_layer::loss(const std::vector<std::vector<double>>& batched_targets) const {

	if (forward_output == nullptr) {
		std::cerr << "Error: No forwad output in dense_layer to calculate loss" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Incompatible batch size, cannot calculate loss" << std::endl;
		exit(EXIT_FAILURE);
	}

	double result = 0.0;
	double* host_forward_output = (double*)malloc(batch_size * neurons * sizeof(double));

	if (host_forward_output == nullptr) {
		std::cerr << "Error: Failed to allocate memory in dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	hipError_t error_code = hipMemcpy(host_forward_output, forward_output, batch_size * neurons * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to host failed in dense_layer" << std::endl;
		exit(error_code);
	}

	for (int i = 0; i < batch_size; i++) {

		if (batched_targets[i].size() != neurons) {
			std::cerr << "Error: Invalid input shape" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (int j = 0; j < neurons; j++) {
			result += ((host_forward_output[i * neurons + j] - batched_targets[i][j]) * (host_forward_output[i * neurons + j] - batched_targets[i][j])) / (double)(batch_size * neurons);
		}
	}
	free(host_forward_output);
	return result;
}

double dense_layer::loss(const std::vector<int>& batched_targets) const {

	//ALL of this will be moved to another function when I start renaming things. 

	if (layer_activation_function != activation_functions::Softmax) {
		std::cerr << "Error: Not a classification model" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (forward_output == nullptr) {
		std::cerr << "Error: No forwad output in dense layer to calculate loss" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Incompatible batch size, cannot calculate loss" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	double* host_forward_output = (double*)malloc(batch_size * neurons * sizeof(double));
	if (host_forward_output == nullptr) {
		std::cerr << "Error: Failed to allocate memory in dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	hipError_t error_code = hipMemcpy(host_forward_output, forward_output, batch_size * neurons * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpyDeviceToHost failed in dense_layer" << std::endl;
		exit(error_code);
	}

	double result = 0;

	for (int i = 0; i < batch_size; i++) {

		if (batched_targets[i] >= neurons) {
			std::cerr << "Error: invalid batched_tagets input" << std::endl;
			exit(EXIT_FAILURE);
		}
		host_forward_output[batched_targets[i]] = (host_forward_output[batched_targets[i]] > 1e-7) ? host_forward_output[batched_targets[i]] : 1e-7;
		host_forward_output[batched_targets[i]] = (host_forward_output[batched_targets[i]] < 1 - 1e-7) ? host_forward_output[batched_targets[i]] : 1 - 1e-7;
		result += -std::log(host_forward_output[batched_targets[i]])/(double)(batch_size);   
	}
	free(host_forward_output);
	return result;
}
double dense_layer::loss(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_targets) const {
	
	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	double result = 0.0;
	int idx = 0;

	double* host_forward_output = (double*)malloc(batch_size * neurons * sizeof(double));
	if (host_forward_output == nullptr) {
		std::cerr << "Error: Failed to allocate memory in dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	hipError_t error_code = hipMemcpy(host_forward_output, forward_output, batch_size * neurons * sizeof(double), hipMemcpyDeviceToHost);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to host failed in dense_layer" << std::endl;
		exit(error_code);
	}

	for (int i = 0; i < batch_size; i++) {

		for (int j = 0; j < batched_targets[i].size(); j++) {
			for (int y = 0; y < batched_targets[i][j].size(); y++) {

				if (idx + batched_targets[i][j][y].size() > neurons) {
					std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
					exit(EXIT_FAILURE);
				}

				for (int x = 0; x < batched_targets[i][j][y].size(); x++) {
					result += (host_forward_output[batch_size * neurons + idx] - batched_targets[i][j][y][x]) * (host_forward_output[batch_size * neurons + idx] - batched_targets[i][j][y][x]) / (double)(batch_size * neurons);
					idx++;
				}
			}
		}

		if (idx != neurons) {
			std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
			exit(EXIT_FAILURE);
		}

		idx = 0;
	}
	free(host_forward_output);
	return result;
}

void dense_layer::init_back_propigation(const std::vector<unsigned int>& batched_targets) {

	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	unsigned int* input_arr = nullptr;

	for (int i = 0; i < batch_size; i++) {
		if (batched_targets[i] >= neurons) {
			std::cerr << "Error: Invalid batched_targets inputs" << std::endl;
			exit(EXIT_FAILURE);
		}
	}

	hipError_t error_code = hipMalloc((void**)&input_arr, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(input_arr, batched_targets.data(), batch_size * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpyHostToDevice failed in dense_layer" << std::endl;
		exit(error_code);
	}

	init_back_propigation(input_arr, batch_size);
	hipFree(input_arr);
}

void dense_layer::init_back_propigation(const std::vector<std::vector<double>>& batched_targets) {
	
	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* input_arr = nullptr;
	hipError_t error_code = hipMalloc((void**)&input_arr, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
		exit(error_code);
	}

	for (size_t i = 0; i < batch_size; i++) {
		
		if (batched_targets[i].size() != neurons) {
			std::cerr << "Error: batched targets of invalid shape" << std::endl;
			exit(EXIT_FAILURE);
		}
		
		error_code = hipMemcpy(input_arr + i * neurons, batched_targets[i].data(), neurons * sizeof(double), hipMemcpyHostToDevice);
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMemcpy to device failed in dense_layer" << std::endl;
			exit(error_code);
		}
	}

	init_back_propigation(input_arr, neurons, batch_size);

	hipFree(input_arr);
}

void dense_layer::init_back_propigation(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_targets) {
	
	if (batched_targets.size() != batch_size) {
		std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* input_arr = nullptr;
	hipError_t error_code = hipMalloc((void**)&input_arr, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
		exit(error_code);
	}

	unsigned int current_size = 0;
	
	for (int i = 0; i < batch_size; i++) {

		for (int j = 0; j < batched_targets[i].size(); j++) {

			for (int y = 0; y < batched_targets[i][j].size(); y++) {

				if (current_size + batched_targets[i][j][y].size() > neurons) {
					std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
					exit(EXIT_FAILURE);
				}
				
				memcpy(input_arr + i * neurons +  current_size, batched_targets[i][j][y].data(), batched_targets[i][j][y].size() * sizeof(double));
				error_code = hipMemcpy(input_arr + i * neurons + current_size, batched_targets[i][j][y].data(), batched_targets[i][j][y].size() * sizeof(double), hipMemcpyHostToDevice);
				if (error_code != hipError_t::hipSuccess) {
					std::cerr << "Error: hipMemcpy to device failed in dense_layer" << std::endl;
					exit(error_code);
				}
				current_size += batched_targets[i][j][y].size();
			}
		}

		if (current_size != neurons) {
			std::cerr << "Error: Batched_targets of incompatible input shape" << std::endl;
			exit(EXIT_FAILURE);
		}

		current_size = 0;
	}

	init_back_propigation(input_arr, neurons, batch_size);

	hipFree(input_arr);
}

void dense_layer::init_back_propigation(double* batched_targets, size_t _input_size, size_t _batch_size) {
	
	if (batch_size != _batch_size || _input_size != neurons) {
		std::cerr << "Error: Invalid input size for dense layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	hipError_t error_code;

	if (backward_input == nullptr) {
		error_code = hipMalloc((void**)&backward_input, batch_size * neurons * sizeof(double));
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
			exit(error_code);
		}
	}

	dim3 blocks(neurons/16 + 1, batch_size/16 + 1);
	dim3 threads(16, 16);

	Cuda_Dense_Layer_Init_Back_Propigation<<<blocks, threads>>>(batched_targets, forward_output, backward_input, batch_size, neurons);
	
	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch init back propigation kernal in dense_layer" << std::endl;
		exit(error_code);
	}

	if (layer_activation_function == activation_functions::Sigmoid) {
		Cuda_Sigmoid_Activation_Backward_Pass<<<blocks, threads>>>(backward_input, forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Rectified_Linear) {
		Cuda_Rectified_Linear_Activation_Backward_Pass<<<blocks, threads>>>(backward_input, forward_output, batch_size, neurons);
	}
	else if (layer_activation_function == activation_functions::Softmax) {
		Cuda_Softmax_Activation_Bakcward_Pass<<<blocks, threads>>>(forward_output, backward_input, batch_size, neurons);
	}

	if (layer_activation_function != activation_functions::Linear && (error_code = hipGetLastError()) != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch backward activation function kernal" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: cudaDevice Synchronize failed in dense_layer" << std::endl;
		exit(error_code);
	}
}

void dense_layer::init_back_propigation(unsigned int* batched_targets, size_t _batch_size) {

	if (batch_size != _batch_size) {
		std::cerr << "Error: Invalid input size for dense layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (layer_activation_function != activation_functions::Softmax) {
		std::cerr << "Error: Invalid activation_function for Init_Cross_Catagorial_Loss in dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	hipError_t error_code;

	if (backward_input == nullptr) {
		error_code = hipMalloc((void**)&backward_input, batch_size * neurons * sizeof(double));
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
			exit(error_code);
		}
	}

	dim3 blocks(neurons / 16 + 1, batch_size / 16 + 1);
	dim3 threads(16, 16);

	Cuda_Dense_Layer_Init_Cross_Catigorial_Loss_Back_Propigation<<<blocks, threads>>>(batched_targets, forward_output, backward_input, batch_size, neurons);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to Launch Init_Cross_Catigorial_Loss_Back_Propigation in dense_layer" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed in dense_layer" << std::endl;
		exit(error_code);
	}
}

void dense_layer::backward(const std::vector<std::vector<double>>& batched_inputs) {
	
	if (batch_size != batched_inputs.size()) {
		std::cerr << "Error: Incompatible batch size" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* input_arr = nullptr;
	hipError_t error_code = hipMalloc((void**)&input_arr, batch_size * inputs * sizeof(double));

	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
		exit(error_code);
	}

	for (int i = 0; i < batch_size; i++) {
		if (batched_inputs[i].size() != inputs) {
			std::cerr << "Error: Batched inputs of invalid shape" << std::endl;
			exit(EXIT_FAILURE);
		}

		error_code = hipMemcpy(input_arr + i * inputs, batched_inputs[i].data(), inputs * sizeof(double), hipMemcpyHostToDevice);
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMemcpy to device failed in dense_layer" << std::endl;
			exit(error_code);
		}
	}

	backward(input_arr, inputs, batch_size);
	hipFree(input_arr);
}

void dense_layer::backward(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_inputs) {
	
	if (batched_inputs.size() != batch_size) {
		std::cerr << "Error: Batched_inputs of incompatible input shape" << std::endl;
		exit(EXIT_FAILURE);
	}

	double* input_arr = (double*)malloc(batch_size * inputs * sizeof(double));
	hipError_t error_code = hipMalloc((void**)&input_arr, batch_size * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
		exit(error_code);
	}

	unsigned int current_size = 0;

	for (int i = 0; i < batch_size; i++) {

		for (int j = 0; j < batched_inputs[i].size(); j++) {
			for (int y = 0; y < batched_inputs[i][j].size(); y++) {

				if (current_size + batched_inputs[i][j][y].size() > inputs) {
					std::cerr << "Error: Batched_inputs of incompatible input shape" << std::endl;
					exit(EXIT_FAILURE);
				}

				error_code = hipMemcpy(input_arr + i * inputs + current_size, batched_inputs[i][j][y].data(), batched_inputs[i][j][y].size() * sizeof(double), hipMemcpyHostToDevice);
				if (error_code != hipError_t::hipSuccess) {
					std::cerr << "Error: hipMemcpy to device failed in dense_layer" << std::endl;
					exit(error_code);
				}
				
				current_size += batched_inputs[i][j][y].size();
			}
		}

		if (current_size != inputs) {
			std::cerr << "Error: Batched_inputs of incompatible input shape" << std::endl;
			exit(EXIT_FAILURE);
		}
		current_size = 0;
	}

	backward(input_arr, inputs, batch_size);
	hipFree(input_arr);
}

void dense_layer::backward(double* batched_inputs, size_t _input_size, size_t _batch_size) {

	if (_batch_size != batch_size || _input_size != inputs) {
		std::cerr << "Error: Invalid input size for backward pass" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (backward_input == nullptr) {
		std::cerr << "Error: Dense_layer not intialized for backward pass" << std::endl;
		exit(EXIT_FAILURE);
	}
	
	hipError_t error_code;
	dim3 blocks(inputs/16 + 1, neurons/16 + 1);
	dim3 threads(16, 16);
	
	Cuda_Dense_Layer_First_Backward_Pass<<<blocks, threads>>>(batched_inputs, backward_input, d_weights, batch_size, neurons, inputs);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: First backward pass kernal failed to launch" << std::endl;
		exit(error_code);
	}
	
	Cuda_Dense_Layer_Second_Backward_Pass<<<neurons / 16 + 1, 16>>>(backward_input, d_bias, batch_size, neurons);

	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Second backward pass kernal failed to launch" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}

}

void dense_layer::backward(layer* prev_layer) {
	
	if (prev_layer->batch_size != batch_size || prev_layer->neurons != inputs) {
		std::cerr << "Error: Prev_layer of invalid input shape or batch size to connect to dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	backward(prev_layer->forward_output, prev_layer->neurons, batch_size);
	
	hipError_t error_code;
	dim3 blocks(inputs / 16 + 1, batch_size / 16 + 1);
	dim3 threads(16, 16);

	if (prev_layer->backward_input == nullptr) {
		
		error_code = hipMalloc((void**)&prev_layer->backward_input, batch_size * inputs * sizeof(double));
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
			exit(error_code);
		}

		error_code = hipMemset(prev_layer->backward_input, 0, batch_size * inputs * sizeof(double));
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMemset failed in dense_layer" << std::endl;
			exit(error_code);
		}

		error_code = hipDeviceSynchronize();
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipDeviceSynchronize failed in dense_layer" << std::endl;
			exit(error_code);
		}
	}

	Cuda_Partial_Derivitive_of_Loss<<<blocks, threads>>>(backward_input, weights, prev_layer->backward_input, batch_size, inputs, neurons);
	
	error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to Launch Partial Derivitive of Loss Kernal" << std::endl;
		exit(error_code);
	}

	if (prev_layer->layer_activation_function == activation_functions::Sigmoid) {
		Cuda_Sigmoid_Activation_Backward_Pass<<<blocks, threads>>>(prev_layer->backward_input, prev_layer->forward_output, batch_size, inputs);
	}
	else if (prev_layer->layer_activation_function == activation_functions::Rectified_Linear) {
		Cuda_Rectified_Linear_Activation_Backward_Pass<<<blocks, threads>>>(prev_layer->backward_input, prev_layer->forward_output, batch_size, inputs);
	}
	else if (prev_layer->layer_activation_function == activation_functions::Softmax) {
		Cuda_Softmax_Activation_Bakcward_Pass<<<blocks, threads>>>(prev_layer->forward_output, prev_layer->backward_input, batch_size, inputs);
	}

	if (prev_layer->layer_activation_function != activation_functions::Linear && (error_code = hipGetLastError()) != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch activation function backward pass kernals" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}
}

void dense_layer::backward(layer* prev_layer, layer* residual_layer) {

	if (prev_layer->neurons != inputs || residual_layer->neurons != neurons || prev_layer->batch_size != batch_size || residual_layer->batch_size != batch_size) {
		std::cerr << "Error: Prev_layer_1 or residual_layer of invalid input shape or batch size to connect to dense_layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	hipError_t error_code;

	if (residual_layer->backward_input == nullptr) {
		error_code = hipMalloc((void**)&residual_layer->backward_input, batch_size * inputs * sizeof(double));
		if (error_code != hipError_t::hipSuccess) {
			std::cerr << "Error: hipMalloc failed in dense_layer" << std::endl;
			exit(error_code);
		}
	}

	error_code = hipMemcpy(residual_layer->backward_input, backward_input, batch_size * neurons * sizeof(double), hipMemcpyDeviceToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy device to device failed in dense_layer" << std::endl;
		exit(error_code);
	}
	
	error_code = hipDeviceSynchronize();
	if (error_code != hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed in dense_layer" << std::endl;
		exit(error_code);
	}

	backward(prev_layer);
}

void dense_layer::update_paramters(double learning_rate) {
	
	dim3 blocks(inputs/16 + 1, neurons/16 + 1);
	dim3 threads(16, 16);

	Cuda_Graident_Decent<<<blocks, threads>>>(d_weights, d_bias, weights, bias, learning_rate, neurons, inputs);

	hipError_t error_code = hipGetLastError();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: Failed to launch Graident Decent kernal in dense_layer" << std::endl;
		exit(error_code);
	}

	error_code = hipDeviceSynchronize();
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipDeviceSynchronize failed" << std::endl;
		exit(error_code);
	}
}

//debugging function will delete later. 
void Print_Cuda_Forward_Output(double* input_arr, size_t batch_size, size_t neurons) {

	double* host_input_arr = (double*)malloc(batch_size * neurons * sizeof(double));
	hipMemcpy(host_input_arr, input_arr, batch_size * neurons * sizeof(double), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < batch_size; i++) {

		std::cout << "{";
		for (int j = 0; j < neurons; j++) {
			std::cout << i * neurons + j << " = > " << host_input_arr[i * neurons + j];
			if (j + 1 < neurons) std::cout << ", ";
		}
		std::cout << "}\n";
	}
	std::cout << "\n";
}