#include "hip/hip_runtime.h"
#include ""
#include "layer.hpp"

#include <iostream>


max_pooling_layer::max_pooling_layer() {
	batch_size = 0;
	input_size = 0;
	channels = 0;
	kernal_size = 0;
	stride = 0;
	inputs = 0;
	output_size = 0;
	neurons = 0;

	forward_output = nullptr;
	backward_input = nullptr;
	layer_activation_function = activation_functions::Linear;
}

max_pooling_layer::max_pooling_layer(size_t _input_size, size_t _channels, size_t _kernal_size, size_t _stride) {

	batch_size = 0;
	input_size = _input_size;
	channels = _channels;
	kernal_size = _kernal_size;
	stride = _stride;

	inputs = channels * input_size * input_size;
	output_size = (input_size - kernal_size) / stride + 1;
	neurons = output_size * output_size * channels;

	forward_output = nullptr;
	backward_input = nullptr;
	layer_activation_function = activation_functions::Linear;
}

max_pooling_layer::~max_pooling_layer() {
	free(forward_output);
	free(backward_input);
}

void max_pooling_layer::forward(const std::vector<std::vector<double>>& batched_inputs) {

	double* input_arr = (double*)malloc(batched_inputs.size() * inputs * sizeof(double));

	if (input_arr == nullptr) {
		std::cerr << "Error: Unable to allocated memory in max_pooling layer for forward pass" << std::endl;
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < batched_inputs.size(); i++) {

		if (batched_inputs[i].size() != inputs) {
			std::cerr << "Error: Batched_inputs of invalid input shape for max_pooling layer" << std::endl;
			exit(EXIT_FAILURE);
		}
		
		memcpy(input_arr + i * inputs, batched_inputs[i].data(), inputs * sizeof(double));
	}

	forward(input_arr, inputs, batched_inputs.size());
	free(input_arr);
}
void max_pooling_layer::forward(const std::vector<std::vector<std::vector<std::vector<double>>>>& batched_inputs) {

	double* input_arr = (double*)malloc(batched_inputs.size() * channels * input_size * input_size * sizeof(double));

	if (input_arr == nullptr) {
		std::cerr << "Error: Unable to allocated memory in max_pooling layer for forward pass" << std::endl;
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < batched_inputs.size(); i++) {

		if (batched_inputs[i].size() != channels) {
			std::cerr << "Error: Batched_inputs of invalid input shape for max_pooling layer" << std::endl;
			exit(EXIT_FAILURE);
		}

		for (int j = 0; j < channels; j++) {

			if (batched_inputs[i][j].size() != input_size) {
				std::cerr << "Error: Batched_inputs of invalid input shape for max_pooling layer" << std::endl;
				exit(EXIT_FAILURE);
			}

			for (int y = 0; y < input_size; y++) {

				if (batched_inputs[i][j][y].size() != input_size) {
					std::cerr << "Error: Batched_inputs of invalid input shape for max_pooling layer" << std::endl;
					exit(EXIT_FAILURE);
				}

				memcpy(input_arr + i * inputs + j * input_size * input_size + y * input_size, batched_inputs[i][j][y].data(), input_size * sizeof(double));
			}
		}
	}
	forward(input_arr, inputs, batched_inputs.size());
	free(input_arr);
}

void max_pooling_layer::forward(double* batched_inputs, size_t _input_size, size_t _batch_size) {

	if (_input_size != inputs) {
		std::cerr << "Error: Batched_inputs of invalid input shape for max_pooling layer" << std::endl;
		exit(EXIT_FAILURE);
	}

	if (_batch_size != batch_size) {

		if (forward_output != nullptr) free(forward_output);
		if (backward_input != nullptr) free(backward_input);

		forward_output = (double*)malloc(_batch_size * inputs * sizeof(double));

		if (forward_output == nullptr) {
			std::cerr << "Error: Unable to allocate memory in max_pooling_layer for forward pass" << std::endl;
			exit(EXIT_FAILURE);
		}

		batch_size = _batch_size;
	}

	double* cuda_batched_input = nullptr;
	double* cuda_forward_output = nullptr;
	hipError_t error_code;

	error_code = hipMalloc((void**)&cuda_batched_input, batch_size * inputs * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}
	
	error_code = hipMalloc((void**)&cuda_forward_output, batch_size * neurons * sizeof(double));
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMalloc failed" << std::endl;
		exit(error_code);
	}

	error_code = hipMemcpy(cuda_batched_input, batched_inputs, batch_size * inputs * sizeof(double), hipMemcpyHostToDevice);
	if (error_code != hipError_t::hipSuccess) {
		std::cerr << "Error: hipMemcpy to device failed" << std::endl;
		exit(error_code);
	}

	//kernal call goes here.

	
}
void max_pooling_layer::forward(const layer* prev_layer) {
	forward(prev_layer->forward_output, prev_layer->neurons, prev_layer->batch_size);
}


